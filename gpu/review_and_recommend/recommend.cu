#include <iostream>
#include <algorithm>
#include <string>
#include <vector>
#include <fstream>
#include <cstdlib>
#include "review_and_recommend.h"
#include "utils.h"

using std::cout;
using std::endl;
using std::string;


int main(int argc, char **argv) {
	if (argc < 5) {
		cout << "Input: ./exe base_file mapping_file num_recs source_words..." << endl;
		return 1;
	}
	
	string base_filename (argv[1]);
	string beg_file = base_filename + "_beg_pos.bin";
	string csr_file = base_filename + "_csr.bin";
	string weight_file = base_filename + "_weight.bin";

	const char *mapping_file = argv[2];
	int num_recs = atoi(argv[3]);
	
	graph<long, long, double, long, long, double> *csr = 
		new graph <long, long, double, long, long, double>
		(beg_file.c_str(), csr_file.c_str(), weight_file.c_str());

	std::cout << "Edges: " << csr->edge_count << std::endl;
    std::cout << "Verticies: " << csr->vert_count << std::endl;

	int num_source_words = argc - 4;
	std::vector<int> source_word_idxs;
	std::vector<string> words = Utils::get_word_mapping(mapping_file);

    for (int i = 0; i < num_source_words; i++) {
		const char *source_word = argv[i + 4];
		auto it = std::find(words.begin(), words.end(), source_word);
		if (it == words.end()) {
			cout << "Not found in graph: " << source_word << endl;
			return 1;
		}
		int idx = std::distance(words.begin(), it);
		source_word_idxs.push_back(idx);
    }

    // Copy source word idxs to device arr
    // Copy csr beg_pos arry into device arry
    // Copy csr csr arry into device arry
    // Copy csr weight arry into device arry
    //Crate closest_words of size num_rec
    // Pass in beg_pos, csr.csr, weight arrays, source_word_idxs arrays, num_recs, and vert_count, closest_words

    // Copy back closest_words

    WordDist **closest_words;
    int *source_idxs_d, *beg_pos_d, *csr_d, *weight_d;
    WordDist **closest_words_d;
    int *num_recs_d;

    HANDLE_ERR(hipMalloc((void **) &source_idxs_d, sizeof(int) * source_word_idxs.size()));
    HANDLE_ERR(hipMalloc((void **) &beg_pos_d, sizeof(int) * csr->vert_count));
    HANDLE_ERR(hipMalloc((void **) &csr_d, sizeof(int) * csr->edge_count));
    HANDLE_ERR(hipMalloc((void **) &weight_d, sizeof(int) * csr->edge_count));
    HANDLE_ERR(hipMalloc((void **) &closest_words_d, sizeof(WordDist*) * num_recs));
    HANDLE_ERR(hipMalloc((void **) &num_recs_d, sizeof(int)));

    HANDLE_ERR(hipMemcpy (source_idxs_d, source_words_idxs, sizeof(int) * source_words_idxs.size(), hipMemcpyHostToDevice));
    HANDLE_ERR(hipMemcpy (beg_pos_d, csr->beg_pos, sizeof(int) * csr->vert_count, hipMemcpyHostToDevice));
    HANDLE_ERR(hipMemcpy (csr_d, csr->csr, sizeof(int) * csr->edge_count, hipMemcpyHostToDevice));
    HANDLE_ERR(hipMemcpy (weight_d, csr->weight, sizeof(int) * csr->edge_count, hipMemcpyHostToDevice));
    HANDLE_ERR(hipMemcpy (num_recs_d, num_recs, sizeof(int), hipMemcpyHostToDevice));

	ReviewAndRec::recommend_kernel(beg_pos_d, csr_d, weight_d, source_idxs_d, num_recs, csr->vert_count, num_recs_d);

    HANDLE_ERR(hipMemcpy (num_recs, num_recs_d, sizeof(int), hipMemcpyDeviceToHost));
    HANDLE_ERR(hipMemcpy (closest_words, closest_words_d, sizeof(WordDist*) * num_recs, hipMemcpyDeviceToHost));

	cout << "\nLearning recommendations :" << endl;
	for (int i = 0; i < num_recs; i++) {
		cout << words[closest_words[i]->word_id] << " (Value: "
			<< closest_words[i]->dist << ")" << endl;
	}
	return 0;	
}
