#include "hip/hip_runtime.h"
#include <sstream>
#include <fstream>
#include <string>
#include <cstdlib>
#include <iostream>
#include <cctype>
#include <vector>
#include <tuple>
#include <cmath>
#include <unordered_map>
#include <unordered_set>
#include "wtime.h"

using std::cout;
using std::endl;
using std::string;

const string ofname = "edge-list.txt";
const string woname = "word-order.txt";

__device__ euclidean_dist(double *x, double *y, int dims) {
    double sum = 0;
    for (unsigned int i = 0; i < dims; i++) {
        double diff = x[i] - y[i];
        sum += diff * diff;
    }
    return sqrt(sum);
}

__global__ void create_edge_list_kernel(double *word_vecs, size_t word_vecs_len, double threshold, 
        double *edge_list, int wv_dims, int *threadwork, size_t *edge_list_size) {
    int wv_num_cols = wv_dims + 1;

    int ie_max_size = word_vecs_len * word_vecs_len / blockDim.x * 3;
    double *inner_edge_list = (int *)malloc(sizeof(double) * ie_max_size);
    int ie_idx = 0;
 
    const int my_thread_id = threadIdx.x + blockIdx.x * blockDim.x;
    const int thread_count = blockDim.x * gridDim.x;

    int tid = my_thread_id;
    while (tid < word_vecs_len) {
        for (unsigned int j = 0; j < word_vecs_len; j++) {
            // No edges to self
            if (tid != j) { 
                double dist = euclidean_dist(word_vecs[tid * wv_num_cols + 1], word_vecs[j * wv_num_cols + 1], wv_dims); 
                if (!threshold || dist < threshold) {
                    double vtx1 = word_vecs[tid * wv_num_cols];
                    double vtx2 = word_vecs[j * wv_num_cols];
                    
                    int ie_base = ie_idx * 3;
                    inner_edge_list[ie_base] = vtx1;
                    inner_edge_list[ie_base + 1] = vtx2;
                    inner_edge_list[ie_base + 2] = dist;
                    ie_idx++;
                }
            }
        }
        tid += thread_count;
    }
    // Add size of inner edge list
    threadwork[my_thread_id + 1] = ie_idx + 1;

    // Single thread aggregates work
    if (threadIdx.x == 1) {
        for (int i = 1; i <= thread_count; i++) {
            threadwork[i] += threadwork[i - 1];
        }
    }

    // Give size back to caller
    *edge_list_size = threadwork[thread_count];

    // Copy thread's work to edge_list
    int base = thread_work[my_thread_id];
    for (int i = 0; i < thread_work[my_thread_id]; i++) {
        int el_row = (base + i) * 3;
        int ie_row = i * 3;
        for (int j = 0; j < 3; j++) {
            edge_list[el_row + j] = inner_edge_list[ie_row + j];
        }
    }
};

double *parse_word_vectors(const char *vecfilename, int limit, int dims, size_t word_size) {
    std::ifstream infile(vecfilename);
    string line;
    int i = 0;
    while (std::getline(infile, line))
        i++;

    double *words = (double *)malloc(sizeof(double) * i * (dims + 1));

    infile.clear();
    infile.seekg(0, std::ios::beg);

	std::ofstream outfile(ofname);
    int row = 0;

    //only read until limit lines
    int limit_counter = 0;
    while (std::getline(infile, line)) {
        if (limit && ++limit_counter > limit) {
            break;
        }
        std::istringstream iss(line);
        string word;
        iss >> word;
        // only take alphanumeric 
        bool alphanum = true;
        for (auto const& ch: word) {
            if (!isalpha(ch)) {
                alphanum = false;
                break;
            }
        }
        if (!alphanum)
            continue;
        // Write to word-order
        outfile << word << endl;

        int base = row * 51;
        words[base] = row;
        double d;
        int d_i = 0;
        while ((iss >> d)) {
            words[base + d_i] = d;
        }
        row++;
    }
    *word_size = row + 1;
    return words;
}

void write_edge_list(double *edge_list, size_t size) {
	std::ofstream outfile(ofname);
	for (unsigned int i = 0; i < size; i++) {
        base = i * 3;
		outfile << edge_list[base] << " " 
			<< edge_list[base + 1] << " " << edge_list[base + 2] << endl;
	}
}

int main(int argc, char *argv[]) {
    if (argc < 2) {
        cout << "./exec vecfile outfile(optional) "
			<< "threshold(optional) limit(optional) "
			<< "to_nums(optional)" << endl;
        return 1;
    }
    const char *vecfilename = argv[1];
    const int write_file = argc > 2 ? atoi(argv[2]) : 0;
    const double threshold = argc > 3 ? atof(argv[3]) : 0;
    const int limit = argc > 4 ? atoi(argv[4]) : 0;
	const int to_nums = argc > 5 ? atoi(argv[5]) : 0;

    size_t word_size;
    double *words = parse_word_vectors(vecfilename, limit, 50, &word_size);

	cout << "Word vec size: " << word_size << endl;

	double *words_d;
	HANDLE_ERR(hipMalloc((void **) &words_d, sizeof(double) * word_size * 51));
	double *edge_list_d;
	HANDLE_ERR(hipMalloc((void **) &edge_list, sizeof(double) * word_size * word_size * 3));
	double *threadwork_d;
	HANDLE_ERR(hipMalloc((void **) &threadwork_d, sizeof(int) * 128 * 128));

    HANDLE_ERR(hipMemcpy (words_d, words, sizeof (double) * word_size * 51, hipMemcpyHostToDevice));

    double starttime = wtime();
    create_edge_list_kernel <<< 128, 128 >>> (words_d, word_size, threshold, edge_list_d, 50, threadwork_d, edge_list_size);
    double endtime = wtime() - starttime;

	double *edge_list = (double *)malloc(sizeof(double) * 3 * edge_list_size);

    HANDLE_ERR(hipMemcpy (edge_list, edge_list_d, sizeof(double) * 3 * edge_list_size, hipMemcpyDeviceToHost));

    write_edge_list(edge_list, edge_list_size);

    cout << "Num edges: " << edge_list.size() << endl;
    cout << "Time: " << endtime << endl;
}
