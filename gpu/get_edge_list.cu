#include "hip/hip_runtime.h"
#include <sstream>
#include <fstream>
#include <string>
#include <cstdlib>
#include <iostream>
#include <cctype>
#include <vector>
#include <tuple>
#include <cmath>
#include <unordered_map>
#include <unordered_set>
#include "error_handler.h"
#include "wtime.h"

using std::cout;
using std::endl;
using std::string;

const string ofname = "edge-list.txt";
const string woname = "word-order.txt";

double __device__ euclidean_dist(double *x, double *y, int dims) {
    double sum = 0;
    for (unsigned int i = 0; i < dims; i++) {
        double diff = x[i] - y[i];
        sum += diff * diff;
    }
    return sqrt(sum);
}

__global__ void create_edge_list_kernel(double *word_vecs, size_t word_vecs_len, 
        double threshold, double *edge_list, int wv_dims) {
    int wv_num_cols = wv_dims + 1;
 
    const int my_thread_id = threadIdx.x + blockIdx.x * blockDim.x;
    const int thread_count = blockDim.x * gridDim.x;

    int tid = my_thread_id;
    while (tid < word_vecs_len) {
        for (unsigned int j = 0; j < word_vecs_len; j++) {
            int write_idx = (tid * word_vecs_len * 3) + (j * 3);
            // No edges to self
            if (tid != j) { 
                double dist = euclidean_dist(word_vecs + 
                        (tid * wv_num_cols + 1), 
                        word_vecs + (j * wv_num_cols + 1), wv_dims); 
                if (!threshold || dist < threshold) {
                    double vtx1 = word_vecs[tid * wv_num_cols];
                    double vtx2 = word_vecs[j * wv_num_cols];
                    edge_list[write_idx] = vtx1;
                    edge_list[write_idx + 1] = vtx2;
                    edge_list[write_idx + 2] = dist;
                } else {
                    edge_list[write_idx] = -1;
                }
            } else {
                edge_list[write_idx] = -1;
            }
        }
        tid += thread_count;
    }
};

double *parse_word_vectors(const char *vecfilename, int limit, int dims, size_t *word_size) {
    std::ifstream infile(vecfilename);
    string line;
    int i = 0;
    while (std::getline(infile, line))
        i++;

    double *words = (double *)malloc(sizeof(double) * i * (dims + 1));

    infile.clear();
    infile.seekg(0, std::ios::beg);

	std::ofstream outfile(woname);
    int row = 0;

    //only read until limit lines
    int limit_counter = 0;
    while (std::getline(infile, line)) {
        if (limit && ++limit_counter > limit) {
            break;
        }
        std::istringstream iss(line);
        string word;
        iss >> word;
        // only take alphanumeric 
        bool alphanum = true;
        for (auto const& ch: word) {
            if (!isalpha(ch)) {
                alphanum = false;
                break;
            }
        }
        if (!alphanum)
            continue;
        // Write to word-order
        outfile << word << endl;

        int base = row * 51;
        words[base] = row;
        double d;
        int d_i = 1;
        while ((iss >> d)) {
            words[base + d_i] = d;
            d_i++;
        }
        row++;
    }
    *word_size = row;
    return words;
}

int write_edge_list(double *edge_list, size_t size) {
	std::ofstream outfile(ofname);
    int counter = 0;
	for (unsigned int i = 0; i < size; i++) {
        int base = i * 3;
        if (edge_list[base] == -1)
            continue;
		outfile << edge_list[base] << " " 
			<< edge_list[base + 1] << " " << edge_list[base + 2] << endl;
        counter++;
	}
    return counter;
}

int main(int argc, char *argv[]) {
    if (argc < 2) {
        cout << "./exec vecfile outfile(optional) "
			<< "threshold(optional) limit(optional) "
			<< "to_nums(optional)" << endl;
        return 1;
    }
    const char *vecfilename = argv[1];
    const int write_file = argc > 2 ? atoi(argv[2]) : 0;
    const double threshold = argc > 3 ? atof(argv[3]) : 0;
    const int limit = argc > 4 ? atoi(argv[4]) : 0;
	const int to_nums = argc > 5 ? atoi(argv[5]) : 0;

    size_t word_size;
    double *words = parse_word_vectors(vecfilename, limit, 50, &word_size);
    
    size_t edge_list_size = word_size * word_size;

	cout << "Word vec size: " << word_size << endl;

	double *words_d;
	HANDLE_ERR(hipMalloc((void **) &words_d, sizeof(double) * word_size * 51));
	double *edge_list_d;
	HANDLE_ERR(hipMalloc((void **) &edge_list_d, sizeof(double) * edge_list_size *3));

    HANDLE_ERR(hipMemcpy (words_d, words, sizeof(double) * word_size * 51, hipMemcpyHostToDevice));

    double starttime = wtime();
    create_edge_list_kernel <<< 128, 128 >>> (words_d, word_size, threshold, edge_list_d, 50);
    hipDeviceSynchronize();
    double endtime = wtime() - starttime;

	double *edge_list = (double *)malloc(sizeof(double) * 3 * edge_list_size);
    HANDLE_ERR(hipMemcpy(edge_list, edge_list_d, sizeof(double) * 3 * edge_list_size, hipMemcpyDeviceToHost));

    int num_edges = write_edge_list(edge_list, edge_list_size);

    cout << "Num edges: " << num_edges << endl;
    cout << "Time: " << endtime << endl;
}
