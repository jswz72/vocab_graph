
#include <hip/hip_runtime.h>
#include <stdio.h>

int main(int argc, char **argv) {
    int nDevices;
    hipGetDeviceCount(&nDevices);
    for (int i =0; i < nDevices; i++) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        printf("Device name: %s\n", prop.name);
        printf("Capabilities: %d.%d\n", prop.major, prop.minor);
    }
    return 0;
}
